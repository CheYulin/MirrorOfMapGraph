#include "hip/hip_runtime.h"
/*********************************************************

Licensed to the Apache Software Foundation (ASF) under one
or more contributor license agreements.  See the NOTICE file
distributed with this work for additional information
regarding copyright ownership.  The ASF licenses this file
to you under the Apache License, Version 2.0 (the
"License"); you may not use this file except in compliance
with the License.  You may obtain a copy of the License at

http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied.  See the License for the
specific language governing permissions and limitations
under the License.

**********************************************************/

/* Written by Erich Elsen and Vishal Vaidyanathan
   of Royal Caliber, LLC
   Contact us at: info@royal-caliber.com
*/

typedef unsigned int uint;

#include "GASEngine.h"
#include "bfs.h"
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/random/normal_distribution.h>
#include <thrust/random/uniform_int_distribution.h>
#include "graphio.h"
#include <iostream>

void generateRandomGraph(std::vector<int> &h_edge_src_vertex,
                         std::vector<int> &h_edge_dst_vertex,
                         int numVertices, int avgEdgesPerVertex) {
  thrust::minstd_rand rng;
  thrust::random::experimental::normal_distribution<float> n_dist(avgEdgesPerVertex, sqrtf(avgEdgesPerVertex));
  thrust::uniform_int_distribution<int> u_dist(0, numVertices - 1);

  for (int v = 0; v < numVertices; ++v) {
    int numEdges = min(max((int)roundf(n_dist(rng)), 1), 1000);
    for (int e = 0; e < numEdges; ++e) {
      uint dst_v = u_dist(rng);
      h_edge_src_vertex.push_back(v);
      h_edge_dst_vertex.push_back(dst_v);
    }
  }
}

int main(int argc, char **argv) {

  int numVertices;
  const char* outFileName = 0;

  //generate simple random graph
  std::vector<int> h_edge_src_vertex;
  std::vector<int> h_edge_dst_vertex;

  if (argc == 1) {
    numVertices = 8000;
    const int avgEdgesPerVertex = 10;
    generateRandomGraph(h_edge_src_vertex, h_edge_dst_vertex, numVertices, avgEdgesPerVertex);
  }
  else if (argc == 2 || argc == 3) {
    loadGraph( argv[1], numVertices, h_edge_src_vertex, h_edge_dst_vertex );
    if (argc == 3)
      outFileName = argv[2];
  }
  else {
    std::cerr << "Too many arguments!" << std::endl;
    exit(1);
  }

  const uint numEdges = h_edge_src_vertex.size();

  thrust::device_vector<int> d_edge_src_vertex = h_edge_src_vertex;
  thrust::device_vector<int> d_edge_dst_vertex = h_edge_dst_vertex;

  //use PSW ordering
  //thrust::sort_by_key(d_edge_dst_vertex.begin(), d_edge_dst_vertex.end(), d_edge_src_vertex.begin());
  thrust::sort_by_key(d_edge_src_vertex.begin(), d_edge_src_vertex.end(), d_edge_dst_vertex.begin());

  thrust::device_vector<int> d_vertex_vals(numVertices, -1);

  std::vector<thrust::device_vector<int> > d_active_vertex_flags;
  {
    thrust::device_vector<int> foo;
    d_active_vertex_flags.push_back(foo);
    d_active_vertex_flags.push_back(foo);
  }
  d_active_vertex_flags[0].resize(numVertices, 0);
  d_active_vertex_flags[1].resize(numVertices, 0);

  //set starting node for bfs
  int startVertex = 0;
  std::vector<char> existing_vertices(numVertices, 0);
  {
    std::vector<int> h_out_edges(numVertices);
    for(int e = 0; e < h_edge_src_vertex.size(); ++e) {
      h_out_edges[h_edge_src_vertex[e]]++;
      existing_vertices[h_edge_src_vertex[e]] = 1;
      existing_vertices[h_edge_dst_vertex[e]] = 1;
    }
    startVertex = std::max_element(h_out_edges.begin(), h_out_edges.end()) - h_out_edges.begin();
  }

  d_vertex_vals[startVertex] = 0;
  d_active_vertex_flags[0][startVertex] = 1;

  GASEngine<bfs, int, int, int, int> engine;

  hipEvent_t start, stop;
  hipEventCreate(&start); hipEventCreate(&stop);

  hipEventRecord(start);

  int diameter = engine.run(d_edge_dst_vertex,
                            d_edge_src_vertex,
                            d_vertex_vals,
                            d_active_vertex_flags);

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float elapsed;
  hipEventElapsedTime(&elapsed, start, stop);
  std::cout << "Took: " << elapsed << " ms" << std::endl;
  std::cout << "Graph Diameter: " << diameter << std::endl;
  std::cout << "M-Edges / sec: " << numEdges / (elapsed * 1000.f) << std::endl;

  if( outFileName )
  {
    FILE* f = fopen( outFileName, "w" );
    thrust::host_vector<int> h_vertex_vals(numVertices);
    thrust::copy( d_vertex_vals.begin(), d_vertex_vals.end(), h_vertex_vals.begin() );
    for( int i = 0; i < existing_vertices.size(); ++i) {
      if (!existing_vertices[i])
        continue;
      fprintf( f, "%d\t%d\n", i, h_vertex_vals[i] );
    }
    fclose(f);
  }
  
  return 0;
}
