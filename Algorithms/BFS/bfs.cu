#include "hip/hip_runtime.h"
/*
 Copyright (C) SYSTAP, LLC 2006-2014.  All rights reserved.

 Licensed under the Apache License, Version 2.0 (the "License");
 you may not use this file except in compliance with the License.
 You may obtain a copy of the License at

 http://www.apache.org/licenses/LICENSE-2.0

 Unless required by applicable law or agreed to in writing, software
 distributed under the License is distributed on an "AS IS" BASIS,
 WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 See the License for the specific language governing permissions and
 limitations under the License.
 */

typedef unsigned int uint;
#include <stdio.h> 
#include <cstdlib>
#include <algorithm>
#include <iostream>
#include <fstream>
#include <string>
#include <deque>
#include <vector>
#include <bfs.h>
#include <iostream>
#include <omp.h>

#include <config.h>

// Utilities and correctness-checking
//#include <test/b40c_test_util.h>

// Graph construction utils

#include <b40c/graph/builder/market.cuh>
#include <b40c/graph/builder/random.cuh>

#include <GASengine/csr_problem.cuh>
#include <GASengine/enactor_vertex_centric.cuh>

using namespace b40c;
using namespace graph;
using namespace std;

template<typename VertexId, typename Value, typename SizeT>
void CPUBFS(int test_iteration,
    const CsrGraph<VertexId, Value, SizeT> &csr_graph,
    VertexId *source_path, SizeT num_srcs, VertexId* srcs)
{
  // (Re)initialize distances
  for (VertexId i = 0; i < csr_graph.nodes; i++)
  {
    source_path[i] = -1;
  }
//  source_path[src] = 0;
  VertexId search_depth = 0;

  // Initialize queue for managing previously-discovered nodes
  std::deque<VertexId> frontier;
  for (int i = 0; i < num_srcs; i++)
  {
    frontier.push_back(srcs[i]);
    source_path[srcs[i]] = 0;
  }

  double startTime = omp_get_wtime();
  //
  // Perform BFS on CPU
  //
  while (!frontier.empty())
  {
    // Dequeue node from frontier
    VertexId dequeued_node = frontier.front();
    frontier.pop_front();
    VertexId neighbor_dist = source_path[dequeued_node] + 1;

    // Locate adjacency list
    int edges_begin = csr_graph.row_offsets[dequeued_node];
    int edges_end = csr_graph.row_offsets[dequeued_node + 1];

    for (int edge = edges_begin; edge < edges_end; edge++)
    {

      // Lookup neighbor and enqueue if undiscovered
      VertexId neighbor = csr_graph.column_indices[edge];
      if (source_path[neighbor] == -1)
      {
        source_path[neighbor] = neighbor_dist;
        if (search_depth < neighbor_dist)
        {
          search_depth = neighbor_dist;
        }
        frontier.push_back(neighbor);
      }
    }
  }

  double EndTime = omp_get_wtime();

  std::cout << "CPU time took: " << (EndTime - startTime) * 1000 << " ms"
      << std::endl;
  search_depth++;
}

bool cudaInit(int device)
{
  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);

  if (error_id != hipSuccess)
  {
    printf("hipGetDeviceCount returned %d\n-> %s\n", (int) error_id,
        hipGetErrorString(error_id));
    printf("Result = FAIL\n");
    exit (EXIT_FAILURE);
  }

  // This function call returns 0 if there are no CUDA capable devices.
  if (deviceCount == 0)
  {
    printf("There are no available device(s) that support CUDA\n");
    return false;
  }
  else
  {
    printf("Detected %d CUDA Capable device(s)\n", deviceCount);
  }

  int dev, driverVersion = 0, runtimeVersion = 0;

  for (dev = 0; dev < deviceCount; ++dev)
  {
    if (dev == device)
    {
      hipSetDevice(dev);
      hipDeviceProp_t deviceProp;
      hipGetDeviceProperties(&deviceProp, dev);

      printf("Running on this device:");
      printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);

      // Console log
      hipDriverGetVersion(&driverVersion);
      hipRuntimeGetVersion(&runtimeVersion);
      printf(
          "  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n",
          driverVersion / 1000, (driverVersion % 100) / 10,
          runtimeVersion / 1000, (runtimeVersion % 100) / 10);
      printf("  CUDA Capability Major/Minor version number:    %d.%d\n",
          deviceProp.major, deviceProp.minor);

      printf(
          "  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n",
          (float) deviceProp.totalGlobalMem / 1048576.0f,
          (unsigned long long) deviceProp.totalGlobalMem);

      break;
    }
  }

  return true;
}

void correctTest(int nodes, int* reference_labels, int* h_labels)
{
  bool pass = true;
  printf("Correctness testing ...");
  for (int i = 0; i < nodes; i++)
  {
    if (reference_labels[i] != h_labels[i])
    {
      printf("Incorrect value for node %d: CPU value %d, GPU value %d\n",
          i, reference_labels[i], h_labels[i]);
      pass = false;
    }
  }
  if (pass)
    printf("passed\n");
  else
    printf("failed\n");
}

void printUsageAndExit(char *algo_name)
{
  std::cout << "Usage: " << algo_name
      << " [-graph (-g) graph_file] [-output (-o) output_file] [-sources src_file] [-BFS \"variable1=value1 variable2=value2 ... variable3=value3\" -help ] [-c config_file]\n";
  std::cout << "     -help display the command options\n";
  std::cout
      << "     -graph specify a sparse matrix in Matrix Market (.mtx) format\n";
  std::cout << "     -output or -o specify file for output result\n";
  std::cout << "     -sources or -s set starting vertices file\n";
  std::cout << "     -c set the BFS options from the configuration file\n";
  std::cout
      << "     -parameters (-p) set the options.  Options include the following:\n";
  Config::printOptions();

  exit(0);
}

int main(int argc, char **argv)
{

  const char* outFileName = 0;
//  int src[1];
//  bool g_undirected;
  const bool g_stream_from_host = false;
  const bool g_with_value = true;
  const bool g_mark_predecessor = false;
  bool g_verbose = false;
  typedef int VertexId; // Use as the node identifier type
  typedef int Value; // Use as the value type
  typedef int SizeT; // Use as the graph size type
  char* graph_file = NULL;
  CsrGraph<VertexId, Value, SizeT> csr_graph(g_stream_from_host);
  char source_file_name[100] = "";
//  int device = 0;
//  double max_queue_sizing = 1.3;
  Config cfg;

  for (int i = 1; i < argc; i++)
  {
    if (strncmp(argv[i], "-help", 100) == 0) // print the usage information
      printUsageAndExit(argv[0]);
    else if (strncmp(argv[i], "-graph", 100) == 0
        || strncmp(argv[i], "-g", 100) == 0)
    { //input graph
      i++;

      graph_file = argv[i];

    }
    else if (strncmp(argv[i], "-output", 100) == 0
        || strncmp(argv[i], "-o", 100) == 0)
    { //output file name
      i++;
      outFileName = argv[i];
    }

    else if (strncmp(argv[i], "-sources", 100) == 0
        || strncmp(argv[i], "-s", 100) == 0)
    { //the file containing starting vertices
      i++;
      strcpy(source_file_name, argv[i]);
    }

    else if (strncmp(argv[i], "-parameters", 100) == 0
        || strncmp(argv[i], "-p", 100) == 0)
    { //The BFS specific options
      i++;
      cfg.parseParameterString(argv[i]);
    }
    else if (strncmp(argv[i], "-c", 100) == 0)
    { //use a configuration file to specify the BFS options instead of command line
      i++;
      cfg.parseFile(argv[i]);
    }
  }

  if (graph_file == NULL)
  {
    printUsageAndExit(argv[0]);
    exit(1);
  }

  char hostname[1024];
  hostname[1023] = '\0';
  gethostname(hostname, 1023);

  printf("Running on host: %s\n", hostname);

  int directed = cfg.getParameter<int>("directed");
  int origin = cfg.getParameter<int>("origin");
  int iter_num = cfg.getParameter<int>("iter_num");

  if (builder::BuildMarketGraph<g_with_value>(graph_file, csr_graph,
      !directed) != 0)
    exit(1);

//  csr_graph.DisplayGraph();
  int num_srcs = 0;
  int* srcs = NULL;

  const int max_src_num = 1000;

  if (strcmp(source_file_name, ""))
  {
    if (strcmp(source_file_name, "RANDOM") == 0)
    {
      printf("Using random starting vertices!\n");
      num_srcs = cfg.getParameter<int>("num_src");
      srcs = new int[num_srcs];
      printf("Using %d random starting vertices!\n", num_srcs);
      srand (time(NULL));int
      count = 0;
      while (count < num_srcs)
      {
        int tmp_src = rand() % csr_graph.nodes;
        if (csr_graph.row_offsets[tmp_src + 1]
            - csr_graph.row_offsets[tmp_src] > 0)
        {
          srcs[count++] = tmp_src;
        }
      }

    }
    else
    {
      printf("Using source file: %s!\n", source_file_name);
      FILE* src_file;
      if ((src_file = fopen(source_file_name, "r")) == NULL)
      {
        printf("Source file open error!\n");
        exit(0);
      }

      srcs = new int[max_src_num];
      for (num_srcs = 0; num_srcs < max_src_num; num_srcs++)
      {
        if (fscanf(src_file, "%d\n", &srcs[num_srcs]) != EOF)
        {
          if (origin == 1)
            srcs[num_srcs]--; //0-based index
        }
        else
          break;
      }
      printf("number of srcs used: %d\n", num_srcs);
    }

  }
  else
  {
    int src_node = cfg.getParameter<int>("src");
    int origin = cfg.getParameter<int>("origin");
    num_srcs = 1;
    srcs = new int[1];
    srcs[0] = src_node;
    if (origin == 1)
      srcs[0]--;
    printf("Single source vertex: %d\n", srcs[0]);
  }

  bool cudaEnabled = cudaInit(cfg.getParameter<int>("device"));
  VertexId* reference_labels;

  int run_CPU = cfg.getParameter<int>("run_CPU");
  if (strcmp(source_file_name, "") == 0 && run_CPU) //Do correctness test only with single starting vertex
  {
    reference_labels = (VertexId*) malloc(
        sizeof(VertexId) * csr_graph.nodes);
    int test_iteration = 1;
//    int src = cfg.getParameter<int>("src");
//    int origin = cfg.getParameter<int>("origin");
//
//    if (origin == 1)
//      src--;

    CPUBFS(test_iteration, csr_graph, reference_labels, 2, srcs);
    //    return 0;
  }

  if (!cudaEnabled)
    return 0;

// Allocate problem on GPU
  int num_gpus = 1;
  typedef GASengine::CsrProblem<bfs, VertexId, SizeT, Value,
      g_mark_predecessor, g_with_value> CsrProblem;
  CsrProblem csr_problem(cfg);
  if (csr_problem.FromHostProblem(g_stream_from_host, csr_graph.nodes,
      csr_graph.edges, csr_graph.column_indices, csr_graph.row_offsets,
      csr_graph.edge_values, csr_graph.row_indices,
      csr_graph.column_offsets, num_gpus,
      directed))
    exit(1);

  const bool INSTRUMENT = true;

  GASengine::EnactorVertexCentric<INSTRUMENT> vertex_centric(cfg, g_verbose);

  for (int i = 0; i < num_srcs; i++)
  {
    int tmpsrcs[1];
    tmpsrcs[0] = srcs[i];

    hipError_t retval = hipSuccess;

    retval = vertex_centric.EnactIterativeSearch<CsrProblem, bfs>(csr_problem,
        csr_graph.row_offsets, directed, 1, tmpsrcs, iter_num);

    if (retval && (retval != hipErrorInvalidDeviceFunction))
    {
      exit(1);
    }
  }

  Value* h_values = (Value*) malloc(sizeof(Value) * csr_graph.nodes);
  csr_problem.ExtractResults(h_values);

  if (strcmp(source_file_name, "") == 0 && run_CPU)
  {
    correctTest(csr_graph.nodes, reference_labels, h_values);
    free(reference_labels);
  }

  if (outFileName)
  {
    FILE* f = fopen(outFileName, "w");
    for (int i = 0; i < csr_graph.nodes; ++i)
    {
      fprintf(f, "%d\n", h_values[i]);
    }

    fclose(f);
  }
  bfs::freeall(csr_problem.graph_slices[0]->vertex_list, csr_problem.graph_slices[0]->edge_list);

  return 0;
}
