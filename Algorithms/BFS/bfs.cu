#include "hip/hip_runtime.h"
/*
 Copyright (C) SYSTAP, LLC 2006-2014.  All rights reserved.

 Licensed under the Apache License, Version 2.0 (the "License");
 you may not use this file except in compliance with the License.
 You may obtain a copy of the License at

 http://www.apache.org/licenses/LICENSE-2.0

 Unless required by applicable law or agreed to in writing, software
 distributed under the License is distributed on an "AS IS" BASIS,
 WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 See the License for the specific language governing permissions and
 limitations under the License.
 */

typedef unsigned int uint;
#include <stdio.h> 
#include <cstdlib>
#include <algorithm>
#include <iostream>
#include <fstream>
#include <string>
#include <deque>
#include <vector>
#include <bfs.h>
#include <iostream>
#include <omp.h>

#include <config.h>

// Utilities and correctness-checking
//#include <test/b40c_test_util.h>

// Graph construction utils

#include <b40c/graph/builder/market.cuh>
#include <b40c/graph/builder/random.cuh>

#include <b40c/graph/GASengine/csr_problem.cuh>
#include <b40c/graph/GASengine/enactor_vertex_centric.cuh>

using namespace b40c;
using namespace graph;
using namespace std;

template<
    typename VertexId,
    typename Value,
    typename SizeT>
void CPUBFS(
    int test_iteration,
    const CsrGraph<VertexId, Value, SizeT> &csr_graph,
    VertexId *source_path,
    VertexId src)
{
  // (Re)initialize distances
  for (VertexId i = 0; i < csr_graph.nodes; i++)
  {
    source_path[i] = -1;
  }
  source_path[src] = 0;
  VertexId search_depth = 0;

  // Initialize queue for managing previously-discovered nodes
  std::deque<VertexId> frontier;
  frontier.push_back(src);

  double startTime = omp_get_wtime();
  //
  // Perform BFS on CPU
  //
  while (!frontier.empty())
  {
    // Dequeue node from frontier
    VertexId dequeued_node = frontier.front();
    frontier.pop_front();
    VertexId neighbor_dist = source_path[dequeued_node] + 1;

    // Locate adjacency list
    int edges_begin = csr_graph.row_offsets[dequeued_node];
    int edges_end = csr_graph.row_offsets[dequeued_node + 1];

    for (int edge = edges_begin; edge < edges_end; edge++)
    {

      // Lookup neighbor and enqueue if undiscovered
      VertexId neighbor = csr_graph.column_indices[edge];
      if (source_path[neighbor] == -1)
      {
        source_path[neighbor] = neighbor_dist;
        if (search_depth < neighbor_dist)
        {
          search_depth = neighbor_dist;
        }
        frontier.push_back(neighbor);
      }
    }
  }

  double EndTime = omp_get_wtime();

  std::cout << "CPU time took: " << (EndTime - startTime) * 1000 << " ms"
      << std::endl;
  search_depth++;
}

bool cudaInit(int device)
{
  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);

  if (error_id != hipSuccess)
  {
    printf("hipGetDeviceCount returned %d\n-> %s\n", (int) error_id,
        hipGetErrorString(error_id));
    printf("Result = FAIL\n");
    exit (EXIT_FAILURE);
  }

  // This function call returns 0 if there are no CUDA capable devices.
  if (deviceCount == 0)
  {
    printf("There are no available device(s) that support CUDA\n");
    return false;
  }
  else
  {
    printf("Detected %d CUDA Capable device(s)\n", deviceCount);
  }

  int dev, driverVersion = 0, runtimeVersion = 0;

  for (dev = 0; dev < deviceCount; ++dev)
  {
    if (dev == device)
    {
      hipSetDevice(dev);
      hipDeviceProp_t deviceProp;
      hipGetDeviceProperties(&deviceProp, dev);

      printf("Running on this device:");
      printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);

      // Console log
      hipDriverGetVersion(&driverVersion);
      hipRuntimeGetVersion(&runtimeVersion);
      printf(
          "  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n",
          driverVersion / 1000, (driverVersion % 100) / 10,
          runtimeVersion / 1000, (runtimeVersion % 100) / 10);
      printf("  CUDA Capability Major/Minor version number:    %d.%d\n",
          deviceProp.major, deviceProp.minor);

      printf(
          "  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n",
          (float) deviceProp.totalGlobalMem / 1048576.0f,
          (unsigned long long) deviceProp.totalGlobalMem);

      break;
    }
  }

  return true;
}

void correctTest(int nodes, int* reference_labels, int* h_labels)
{
  bool pass = true;
  printf("Correctness testing ...");
  for (int i = 0; i < nodes; i++)
  {
    if (reference_labels[i] != h_labels[i])
    {
      printf("Incorrect value for node %d: CPU value %d, GPU value %d\n", i, reference_labels[i], h_labels[i]);
      pass = false;
    }
  }
  if (pass)
    printf("passed\n");
}

void printUsageAndExit()
{
  std::cout
      << "Usage: ./BFS [-graph (-g) graph_file] [-sources src_file] [-BFS \"variable1=value1 variable2=value2 ... variable3=value3\" -help ] [-c config_file]\n";
  std::cout << "     -help display the command options\n";
  std::cout << "     -graph specify a sparse matrix in Matrix Market (.mtx) format\n";
  std::cout << "     -sources or -s set starting vertices file\n";
  std::cout << "     -c set the BFS options from the configuration file\n";
  std::cout
      << "     -BFS set the options.  Options include the following:\n";
  Config::printOptions();

  exit(0);
}

int main(int argc, char **argv)
{

  const char* outFileName = 0;
//  int src[1];
//  bool g_undirected;
  const bool g_stream_from_host = false;
  const bool g_with_value = true;
  const bool g_mark_predecessor = false;
  bool g_verbose = false;
  typedef int VertexId; // Use as the node identifier type
  typedef int Value; // Use as the value type
  typedef int SizeT; // Use as the graph size type
  char* graph_file = NULL;
  CsrGraph<VertexId, Value, SizeT> csr_graph(g_stream_from_host);
  char source_file_name[100] = "";
//  int device = 0;
//  double max_queue_sizing = 1.3;
  Config cfg;

  for (int i = 1; i < argc; i++)
  {
    if (strncmp(argv[i], "-help", 100) == 0) // print the usage information
      printUsageAndExit();
    else if (strncmp(argv[i], "-graph", 100) == 0
        || strncmp(argv[i], "-g", 100) == 0)
    { //input graph
      i++;

      graph_file = argv[i];

    }
    else if (strncmp(argv[i], "-output", 100) == 0 || strncmp(argv[i], "-o", 100) == 0)
    { //output file name
      i++;
      outFileName = argv[i];
    }

    else if (strncmp(argv[i], "-sources", 100) == 0 || strncmp(argv[i], "-s", 100) == 0)
    { //the file containing starting vertices
      i++;
      strcpy(source_file_name, argv[i]);
    }

    else if (strncmp(argv[i], "-BFS", 100) == 0)
    { //The BFS specific options
      i++;
      cfg.parseParameterString(argv[i]);
    }
    else if (strncmp(argv[i], "-c", 100) == 0)
    { //use a configuration file to specify the BFS options instead of command line
      i++;
      cfg.parseFile(argv[i]);
    }
  }

  if (graph_file == NULL)
  {
    printUsageAndExit();
    exit(1);
  }

  char hostname[1024];
  hostname[1023] = '\0';
  gethostname(hostname, 1023);

  printf("Running on host: %s\n", hostname);

  int directed = cfg.getParameter<int>("directed");

  if (builder::BuildMarketGraph<g_with_value>(graph_file, csr_graph,
      !directed) != 0)
    exit(1);

  bool cudaEnabled = cudaInit(cfg.getParameter<int>("device"));
  VertexId* reference_labels = (VertexId*) malloc(sizeof(VertexId) * csr_graph.nodes);
  if (strcmp(source_file_name, "") == 0)//Do correctness test only with single starting vertex
  {
    int test_iteration = 1;
    int src = cfg.getParameter<int>("src");
    int origin = cfg.getParameter<int>("origin");

    if(origin == 1)
      src--;

    CPUBFS(
        test_iteration,
        csr_graph,
        reference_labels,
        src);
    //    return 0;
  }

  if (!cudaEnabled)
    return 0;

  VertexId* h_labels = (VertexId*) malloc(sizeof(VertexId) * csr_graph.nodes);
  int* h_dists = (VertexId*) malloc(sizeof(VertexId) * csr_graph.nodes);
//    VertexId* reference_check = (g_quick) ? NULL : reference_labels;
//
//    //Allocate host-side node_value array (both ref and gpu-computed results)
//    Value* ref_node_values = (Value*) malloc(sizeof(Value) * csr_graph.nodes);
  Value* h_node_values = (Value*) malloc(sizeof(Value) * csr_graph.nodes);
//    Value* ref_node_value_check = (g_quick) ? NULL : ref_node_values;
//
//    //Allocate host-side sigma value array (both ref and gpu-computed results)
//    Value* ref_sigmas = (Value*) malloc(sizeof(Value) * csr_graph.nodes);
  Value* h_sigmas = (Value*) malloc(sizeof(Value) * csr_graph.nodes);
//    Value* ref_sigmas_check = (g_quick) ? NULL : ref_sigmas;
  Value* h_deltas = (Value*) malloc(sizeof(Value) * csr_graph.nodes);

// Allocate problem on GPU
  int num_gpus = 1;
  typedef GASengine::CsrProblem<bfs, VertexId, SizeT, Value,
      g_mark_predecessor, g_with_value> CsrProblem;
  CsrProblem csr_problem(cfg);
  if (csr_problem.FromHostProblem(source_file_name, g_stream_from_host, csr_graph.nodes,
      csr_graph.edges, csr_graph.column_indices,
      csr_graph.row_offsets, csr_graph.edge_values, csr_graph.row_indices,
      csr_graph.column_offsets, csr_graph.node_values, num_gpus))
    exit(1);

  const bool INSTRUMENT = true;

  GASengine::EnactorVertexCentric<INSTRUMENT> vertex_centric(cfg, g_verbose);

  hipError_t retval = hipSuccess;

  retval = vertex_centric.EnactIterativeSearch<CsrProblem, bfs>(csr_problem, source_file_name,
      csr_graph.row_offsets);

  if (retval && (retval != hipErrorInvalidDeviceFunction))
  {
    exit(1);
  }

  csr_problem.ExtractResults(h_dists, h_labels, h_sigmas, h_deltas);

  if (strcmp(source_file_name, "") == 0)
    correctTest(csr_graph.nodes, reference_labels, h_dists);

  if (outFileName)
  {
    FILE* f = fopen(outFileName, "w");
    for (int i = 0; i < csr_graph.nodes; ++i)
    {
      fprintf(f, "%d\n", h_dists[i]);
    }

    fclose(f);
  }

  return 0;
}
